#include "hip/hip_runtime.h"
#include "blake2.h"
#include "bc_miner.h"

#include <string>
#include <vector>
#include <algorithm>
#include <ctime>

#include <iostream>

/*
INFO	 mining.thread worker 4517 reporting in 
INFO	 mining.primitives twork: 7296c034e95304ee2a69c2a61e6287a0 58448d08d723222d19658c0364dd247c 64
                                  58448d08d723222d19658c0364dd247c
INFO	 mining.primitives miner: 0xf34fa87db39d15471bebe997860dcd49fc259318 42 
INFO	 mining.primitives merkl: 108459d41ce2399e13992528ea1bd9940fac9df181e29c2b61caeaad55f6532a 64 
INFO	 mining.primitives nhash: 1af153f4cf971b61cc867b760bebfbc98a8c216cd69d6bb5dbb63aaed9db1fc3 64 
INFO	 mining.primitives times: 1536810114 10 
INFO	 mining.primitives cocat: 0xf34fa87db39d15471bebe997860dcd49fc259318108459d41ce2399e13992528ea1bd9940fac9df181e29c2b61caeaad55f6532a1af153f4cf971b61cc867b760bebfbc98a8c216cd69d6bb5dbb63aaed9db1fc31536810114 180 
INFO	 mining.primitives solun: 2b08d9146a6ce1f02db88203bdc653cd0b5e33ec945f391f82dbe3d417fb586e 64 
INFO	 mining.primitives wrkck: 53,56,52,52,56,100,48,56,100,55,50,51,50,50,50,100,49,57,54,53,56,99,48,51,54,52,100,100,50,52,55,99, 55,50,57,54,99,48,51,52,101,57,53,51,48,52,101,101,50,97,54,57,99,50,97,54,49,101,54,50,56,55,97,48 2 
INFO	 mining.primitives compr: 2b08d9146a6ce1f02db88203bdc653cd0b5e33ec945f391f82dbe3d417fb586e 64 
INFO	 mining.primitives testr: 204933315567342 undefined
*/

int mypow(int base, int exp) {
  int result = 1;
  while( exp-- ) { result *= base; }
  return result;
}

struct sort_by_distance {
  const size_t* distances;
  bool operator()(size_t i1,size_t i2) const {
    //std::cout << i1 << ' ' << distances[i1] << " >?= " << i2 << ' ' << distances[i2] << std::endl;
    return distances[i1] >= distances[i2]; 
  }
};

int main(int argc, char **argv) {
    
    std::string work ("0edd781347cfc9c3ff49fdc423c7f1a3deae6501e5cef6b99c45c8901f763320");
    std::string mhash("0xf34fa87db39d15471bebe997860dcd49fc259318");
    std::string merkl("7aff5341ec1a1caa51c74c162c7f2a3946fe28f23b6e630de995f74d5767f865");
    uint32_t thenonce = 2060688607;
    uint8_t nonce_string[12]; // ten bytes and a null character max;
    memset(nonce_string,0,12);
    // convert nonce
    static uint16_t num_to_code[16] = {48,49,50,51,52,53,54,55,56,57};
    nonce_string[0] = '0'; // take care of base case
    uint32_t length = 0;
    while( thenonce >= std::pow(10,length) ) { ++length; }
    std::cout << "the length: " << length << std::endl;
    for( uint32_t i = 0; i < length; ++i ){
      nonce_string[length-i-1] = num_to_code[(thenonce/mypow(10,i))%10];
    }
    std::cout << thenonce << ' ' << nonce_string << std::endl;    
    std::string nhash("cb5d17fe5c27f7b7426002eb665142d00190553b9d945a936eed3ffd23cdde71");
    std::string times("1536783719");

    std::string the_thing = mhash + merkl + nhash + times;

    std::string result_bc("c0d42acc9793a81096411b74b78fe9a12645737c57ee1544fb35d5fa6f09503e");
        
    // now let's do it on the GPU for real
    size_t stash_size = mhash.length();
    size_t tstamp_size = times.length();
    

    bc_mining_inputs in;
    bc_mining_outputs out;
    bc_mining_mempools mempool;

    in.miner_key_size_ = mhash.length();
    in.time_stamp_size_ = times.length();
    in.work_size_ = work.length();    
    in.the_difficulty_ = 303810187437540ULL;

    memcpy(in.miner_key_,mhash.c_str(),in.miner_key_size_);
    memcpy(in.merkel_root_,merkl.c_str(),BLAKE2B_OUTBYTES);
    memcpy(in.time_stamp_,times.c_str(),in.time_stamp_size_);
    //set the work
    for(unsigned i = 0; i < in.work_size_; ++i ) {
      char temp[2];
      temp[0] = work[i];
      temp[1] = '\0';
      in.received_work_[i/2] += strtol(temp,NULL,16)<<(4*((i+1)%2));
    }


    init_mining_memory(mempool);

    run_miner(in,mempool,out);
    
    std::cout << "gpu: " << "blep" << " trial = 0x" << std::hex;
    // output "blake2bl"
    for( unsigned i = 32; i < BLAKE2B_OUTBYTES; ++i ) {
    	 std::cout << std::hex << (unsigned)(out.result_blake2b_[i]>>4) << (unsigned)(out.result_blake2b_[i]&0xf);
    }
    std::cout << std::dec << std::endl;
    std::cout << "gpu distance is: " << out.distance_ << std::endl;
    
    destroy_mining_memory(mempool);
    
    return 0;
}
