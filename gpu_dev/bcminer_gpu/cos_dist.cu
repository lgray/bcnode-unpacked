#include "hip/hip_runtime.h"
// cosine distance implementation for blockcollider
// lgray@github September 2018
// permission granted to use under MIT license

#include <iostream>

__host__ __device__ double cosine_distance_cu(uint8_t work[BLAKE2B_OUTBYTES],
                                              uint8_t comp[BLAKE2B_OUTBYTES],
                                              size_t bytes_size=BLAKE2B_OUTBYTES) {

  static uint32_t num_to_code[16] = {48,49,50,51,52,53,54,55,56,57,97,98,99,100,101,102};
  
  double acc(0), num(0), den(0);
  double norm_work_t, norm_work_s, norm_work_r;
  double norm_comp_t, norm_comp_s, norm_comp_r;
  #pragma unroll
  for(unsigned j = 2; j < BLAKE2B_OUTBYTES/16; ++j) {
    uint32_t jwork1(0), jwork2(0), jcomp1(0), jcomp2(0);
    num = 0; den = 0; 
    norm_work_t = 0; norm_work_s = 0; norm_work_r = 0;
    norm_comp_t = 0; norm_comp_s = 0; norm_comp_r = 0;
    #pragma unroll
    for( unsigned i = 0; i < 16; ++i ) {      
      unsigned offset_fwd = 16*j + i;      
      unsigned offset_bkw = (16*(4-j-1)) + i;
      unsigned work_lcl = work[offset_bkw];
      unsigned comp_lcl = comp[offset_fwd];
      jwork2 = num_to_code[work_lcl&0xf];
      jcomp2 = num_to_code[comp_lcl&0xf];
      jwork1 = num_to_code[(work_lcl>>4)&0xf];
      jcomp1 = num_to_code[(comp_lcl>>4)&0xf];
      num += jwork1*jcomp1; num += jwork2*jcomp2;

      bool mask = jwork1 > norm_work_t;
      norm_work_r = mask ? norm_work_t / jwork1 : jwork1 / norm_work_t;
      norm_work_s = mask*(1+norm_work_s*norm_work_r*norm_work_r)+(!mask)*(norm_work_s+norm_work_r*norm_work_r);
      norm_work_t = mask*jwork1 + (!mask)*norm_work_t;

      mask = jwork2 > norm_work_t;
      norm_work_r = mask ? norm_work_t / jwork2 : jwork2 / norm_work_t;
      norm_work_s = mask*(1+norm_work_s*norm_work_r*norm_work_r)+(!mask)*(norm_work_s+norm_work_r*norm_work_r);
      norm_work_t = mask*jwork2 + (!mask)*norm_work_t;

      mask = jcomp1 > norm_comp_t;
      norm_comp_r = mask ? norm_comp_t / jcomp1 : jcomp1 / norm_comp_t;
      norm_comp_s = mask*(1+norm_comp_s*norm_comp_r*norm_comp_r)+(!mask)*(norm_comp_s+norm_comp_r*norm_comp_r);
      norm_comp_t = mask*jcomp1 + (!mask)*norm_comp_t;

      mask = jcomp2 > norm_comp_t;
      norm_comp_r = mask ? norm_comp_t / jcomp2 : jcomp2 / norm_comp_t;
      norm_comp_s = mask*(1+norm_comp_s*norm_comp_r*norm_comp_r)+(!mask)*(norm_comp_s+norm_comp_r*norm_comp_r);
      norm_comp_t = mask*jcomp2 + (!mask)*norm_comp_t;

    }
    den = (norm_work_t*std::sqrt(norm_work_s))*(norm_comp_t*std::sqrt(norm_comp_s));
    acc += (1.0-num/den);
  }  
  return acc*1000000000000000ULL;
}
