#include "hip/hip_runtime.h"
// The basic components of a GPU-based block collider miner
// lgray@github September 2018
// permission granted to use under MIT license
// this is a GPU miner for block collider that does ~ 20M hashes + distances per second

#include "bc_miner.h"
#include "blake2.h"
#include "blake2b.cu"
#include "cos_dist.cu"
#include <hiprand/hiprand_kernel.h>
#include "stdio.h"
#include <pthread.h>

__global__ void setup_rand(hiprandState* state, uint32_t random)
{
  unsigned id = threadIdx.x + blockIdx.x * blockDim.x;
  
  /* Each thread gets same seed, a different sequence 
     number, no offset */
  hiprand_init((1234^(uint64_t)state + id^4321)^random, 0, 0, &state[id]);
}

//__device__ __host__ __forceinline__ 
__global__
void one_unit_work(bc_mining_data* mining_info) {
  
  unsigned id = threadIdx.x + blockIdx.x *blockDim.x;
  
  uint8_t data_in[bc_mining_data::INLENGTH];
  //memset(data_in,0,bc_mining_data::INLENGTH); // this memset is unecessary 
  
  const size_t idoffset = id*BLAKE2B_OUTBYTES;
  memcpy(data_in,mining_info->work_template_,mining_info->work_size_);
  memcpy(data_in+mining_info->nonce_hash_offset_,mining_info->nonce_hashes+idoffset,BLAKE2B_OUTBYTES);

  
  blake2b_state s;
  blake2b_init_cu(&s,BLAKE2B_OUTBYTES);  
  blake2b_update_cu(&s,data_in,mining_info->work_size_);
  blake2b_final_cu(&s,mining_info->result+idoffset,BLAKE2B_OUTBYTES);
  

  mining_info->distance[id] = cosine_distance_cu(mining_info->received_work_,
						 mining_info->result+id*BLAKE2B_OUTBYTES);
}

__global__
void prepare_work_nonces(hiprandState *state, bc_mining_data* mining_info) {

  static uint16_t num_to_code[16] =  {48,49,50,51,52,53,54,55,56,57,97,98,99,100,101,102};  

  static uint64_t powers_of_ten[11] = { 1,
					10,
					100,
					1000,
					10000,
					100000,
					1000000,
					10000000,
					100000000,
					1000000000,
					10000000000};

  unsigned id = threadIdx.x + blockIdx.x * blockDim.x;
    
  hiprandState localState = state[id];
  uint8_t nonce_string[12]; // ten bytes and a null character max;
  uint8_t nonce_hash[BLAKE2B_OUTBYTES];
  memset(nonce_string,0,12);

  //2060688607;
  uint32_t nonce = hiprand(&localState);
  
  // convert nonce
  nonce_string[0] = '0'; // take care of base case
  uint32_t length = 0;
  while( nonce >= powers_of_ten[length] ) { ++length; }  
  for( uint32_t i = 0; i < length; ++i ) {
    nonce_string[length-i-1] = num_to_code[(nonce/powers_of_ten[i])%10];
  }
  length = (length == 0) + (length > 0)*length;
  
  //printf("length: %u %u %s\n",length,nonce,nonce_string); 
  
  // create the nonce hash
  blake2b_state ns;
  blake2b_init_cu(&ns,BLAKE2B_OUTBYTES);  
  blake2b_update_cu(&ns,nonce_string,length);
  blake2b_final_cu(&ns,nonce_hash,BLAKE2B_OUTBYTES);
  
  // convert nonce in place to string codes and "blake2bl" form
  #pragma unroll
  for( unsigned i = 32; i < BLAKE2B_OUTBYTES; ++i ) {
    uint8_t byte = nonce_hash[i];
    nonce_hash[2*(i-32)] = num_to_code[byte>>4];
    nonce_hash[2*(i-32)+1] = num_to_code[byte&0xf];
  }
    
  // now we put everything into the data_in string in stringified hex form  
  const size_t idoffset = id*BLAKE2B_OUTBYTES;
  memcpy(mining_info->nonce_hashes+idoffset,
	 nonce_hash,
	 BLAKE2B_OUTBYTES);  

  //copy the local work back to the gpu memory  
  mining_info->nonce[id] = nonce;

  state[id] = localState;
}

__global__ void prepare_max_distance(uint64_t *max, uint64_t *maxidx, const uint64_t *a) {
  __shared__ uint64_t maxtile[N_MINER_THREADS_PER_BLOCK];
  __shared__ uint64_t maxidxtile[N_MINER_THREADS_PER_BLOCK];
  
  unsigned int tid = threadIdx.x;
  uint64_t i = blockIdx.x * blockDim.x + threadIdx.x;
  maxtile[tid] = a[i];
  maxidxtile[tid] = i;
  __syncthreads();
  
  //sequential addressing by reverse loop and thread-id based indexing
  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s) {
      if (maxtile[tid + s] > maxtile[tid]) {
	maxtile[tid] = maxtile[tid + s];
	maxidxtile[tid] = maxidxtile[tid + s];
      }
    }
    __syncthreads();
  }
  
  if (tid == 0) {
    max[blockIdx.x] = maxtile[0];
    maxidx[blockIdx.x] = maxidxtile[0];
  }
}

__global__ void finalize_max_distance(uint64_t *max, uint64_t *maxidx) {
  __shared__ uint64_t maxtile[N_MINER_THREADS_PER_BLOCK];
  __shared__ uint64_t maxidxtile[N_MINER_THREADS_PER_BLOCK];

  unsigned int tid = threadIdx.x;
  uint64_t i = blockIdx.x * blockDim.x + threadIdx.x;
  maxtile[tid] = max[i];
  maxidxtile[tid] = maxidx[i];
  __syncthreads();
  
  //sequential addressing by reverse loop and thread-id based indexing
  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s) {
      if (maxtile[tid + s] > maxtile[tid]) {
	maxtile[tid] = maxtile[tid + s];
	maxidxtile[tid] = maxidxtile[tid + s];
      }
    }
    __syncthreads();
  }
  
  if (tid == 0) {    
    max[blockIdx.x] = maxtile[0];
    maxidx[blockIdx.x] = maxidxtile[0];
  }
}

void init_gpus(std::vector<bc_mining_stream>& streams) {
  streams.clear();
  int nGPUs = 0;
  hipGetDeviceCount(&nGPUs);
  std::cout << "Found " << nGPUs << " GPUs to use for mining!" << std::endl;

  streams.resize(nGPUs);
  for( unsigned iGPU = 0; iGPU < nGPUs; ++iGPU ) {
    streams[iGPU].device = iGPU;
    hipSetDevice(iGPU);
    hipStreamCreate(&streams[iGPU].stream);
    init_mining_memory(streams[iGPU].pool,streams[iGPU].stream);
  }
}

// create the primary mining work areas
// run this once to create the memory pools necessary for mining
// large cudaMallocs take a long time, hipMemset is fast
void init_mining_memory(bc_mining_mempools& pool, hipStream_t stream) {
  if( pool.dev_cache != NULL ) return;
  if( pool.dev_states != NULL ) return;
  if( pool.scratch_dists != NULL ) return;
  if( pool.scratch_indices != NULL ) return;

  // allocate device memory for random states and hashing work
  hipStreamSynchronize(stream);
  hipMalloc((void **)&pool.dev_states, HASH_TRIES * 1 * sizeof(hiprandState));
  hipMalloc(&pool.dev_cache,sizeof(bc_mining_data));
  hipMalloc(&pool.scratch_dists,HASH_TRIES*sizeof(uint64_t));
  hipMalloc(&pool.scratch_indices,HASH_TRIES*sizeof(uint64_t));
  hipStreamSynchronize(stream);
}

void run_miner(const bc_mining_inputs& in, bc_mining_stream& bcstream, bc_mining_outputs& out) {
  hipSetDevice(bcstream.device);
  hipStream_t stream = bcstream.stream;
  bc_mining_mempools& pool = bcstream.pool;

  if( pool.dev_cache == NULL ) return;
  if( pool.dev_states == NULL ) return;
  if( pool.scratch_dists == NULL ) return;
  if( pool.scratch_indices == NULL ) return;
  
  const unsigned max_iterations = 100;
  
  dim3 threads(N_MINER_THREADS_PER_BLOCK,1,1), blocks(HASH_TRIES/N_MINER_THREADS_PER_BLOCK,1,1);
  
  //random numbers
  uint16_t work_size = in.miner_key_size_ + 2*BLAKE2B_OUTBYTES + in.time_stamp_size_;
  uint16_t nonce_hash_offset = in.miner_key_size_ + BLAKE2B_OUTBYTES;
    
  // prepare the mining work
  hipMemsetAsync(pool.dev_cache,0,sizeof(bc_mining_data),stream);
  hipMemcpyAsync(&pool.dev_cache->time_stamp_size_, &in.time_stamp_size_, sizeof(size_t), hipMemcpyHostToDevice,stream);
  hipMemcpyAsync(pool.dev_cache->time_stamp_, in.time_stamp_, in.time_stamp_size_, hipMemcpyHostToDevice,stream);
  hipMemcpyAsync(&pool.dev_cache->miner_key_size_, &in.miner_key_size_, sizeof(size_t), hipMemcpyHostToDevice,stream);
  hipMemcpyAsync(pool.dev_cache->miner_key_, in.miner_key_, in.miner_key_size_, hipMemcpyHostToDevice,stream);
  hipMemcpyAsync(pool.dev_cache->received_work_, in.received_work_, BLAKE2B_OUTBYTES, hipMemcpyHostToDevice,stream);
  hipMemcpyAsync(pool.dev_cache->merkel_root_,in.merkel_root_, BLAKE2B_OUTBYTES, hipMemcpyHostToDevice,stream);

  //setup the work template
  hipMemsetAsync(pool.dev_cache->work_template_,0,bc_mining_data::INLENGTH,stream);
  hipMemcpyAsync(&pool.dev_cache->nonce_hash_offset_,&nonce_hash_offset,sizeof(uint16_t),hipMemcpyHostToDevice,stream);
  hipMemcpyAsync(&pool.dev_cache->work_size_,&work_size,sizeof(uint16_t),hipMemcpyHostToDevice,stream);
  unsigned index = 0;
  hipMemcpyAsync(pool.dev_cache->work_template_,pool.dev_cache->miner_key_,in.miner_key_size_,hipMemcpyDeviceToDevice,stream);
  index += in.miner_key_size_;
  hipMemcpyAsync(pool.dev_cache->work_template_+index,pool.dev_cache->merkel_root_,BLAKE2B_OUTBYTES,hipMemcpyDeviceToDevice,stream);
  index += 2*BLAKE2B_OUTBYTES; //advance past nonce hash area
  hipMemcpyAsync(pool.dev_cache->work_template_+index,pool.dev_cache->time_stamp_,in.time_stamp_size_,hipMemcpyDeviceToDevice,stream);
  index += in.time_stamp_size_;
  
  // work areas for finding max
  uint64_t max_value(0), max_idx(0);
  hipMemsetAsync(pool.scratch_dists,0,HASH_TRIES*sizeof(uint64_t),stream);
  hipMemsetAsync(pool.scratch_indices,0,HASH_TRIES*sizeof(uint64_t),stream);
  
  uint64_t iterations = 0;
  // the following kernel launches are the primary work
  // only set the random seeds once
  setup_rand<<<blocks,threads,0,stream>>>(pool.dev_states,((const uint32_t*)in.received_work_)[0]);
  do {
    hipMemsetAsync(pool.dev_cache->result,0,HASH_TRIES*BLAKE2B_OUTBYTES,stream);
    hipMemsetAsync(pool.dev_cache->nonce,0,HASH_TRIES*sizeof(uint32_t),stream);
    hipMemsetAsync(pool.dev_cache->nonce_hashes,0,HASH_TRIES*BLAKE2B_OUTBYTES,stream);
    prepare_work_nonces<<<blocks,threads,0,stream>>>(pool.dev_states,pool.dev_cache);
    one_unit_work<<<blocks,threads,0,stream>>>(pool.dev_cache);
    hipMemsetAsync(pool.scratch_dists,0,HASH_TRIES*sizeof(uint64_t),stream);
    hipMemsetAsync(pool.scratch_indices,0,HASH_TRIES*sizeof(uint64_t),stream);
    prepare_max_distance<<<blocks,threads,0,stream>>>(pool.scratch_dists,pool.scratch_indices,pool.dev_cache->distance);
    unsigned temp = blocks.x;
    while( temp > threads.x ) {
      temp /= threads.x;
      finalize_max_distance<<<temp,threads,0,stream>>>(pool.scratch_dists,pool.scratch_indices);
    }
    finalize_max_distance<<<1,temp,0,stream>>>(pool.scratch_dists,pool.scratch_indices);
    // get the max value and index, which are at index zero in the scratch arrays
    hipMemcpyAsync(&max_value,pool.scratch_dists,sizeof(uint64_t),hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(&max_idx,pool.scratch_indices,sizeof(uint64_t),hipMemcpyDeviceToHost,stream);
    const uint64_t offsetb2b = max_idx*BLAKE2B_OUTBYTES;
    hipMemcpyAsync(out.result_blake2b_,pool.dev_cache->result+offsetb2b, BLAKE2B_OUTBYTES,hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(&out.nonce_, &pool.dev_cache->nonce[max_idx], sizeof(uint32_t), hipMemcpyDeviceToHost,stream);
    hipStreamSynchronize(stream);
    ++iterations;
  } while( max_value <= in.the_difficulty_ && iterations <= max_iterations );

  out.difficulty_ = in.the_difficulty_;
  out.distance_ = max_value;
  out.iterations_ = iterations*HASH_TRIES;
}

void* run_miner_thread(void * input) {
  bc_thread_data& inputs = *((bc_thread_data*)input);
  run_miner(*inputs.in,*inputs.stream,*inputs.out);
  return NULL;
}

void destroy_mining_memory(bc_mining_mempools& pool, hipStream_t stream) {
  if( pool.dev_cache == NULL ) return;
  if( pool.dev_states == NULL ) return;
  if( pool.scratch_dists == NULL ) return;
  if( pool.scratch_indices == NULL ) return;

  // free device memory
  hipStreamSynchronize(stream);
  hipFree(pool.dev_states);
  hipFree(pool.dev_cache);
  hipFree(pool.scratch_dists);
  hipFree(pool.scratch_indices);
  hipStreamSynchronize(stream);

  // set it to null
  pool.dev_states = NULL;
  pool.dev_cache = NULL;
  pool.scratch_dists = NULL;
  pool.scratch_indices = NULL;
}

void destroy_gpus(std::vector<bc_mining_stream>& streams) {
  for(unsigned i = 0; i < streams.size(); ++i ) {
    hipSetDevice(streams[i].device);
    destroy_mining_memory(streams[i].pool,streams[i].stream);
    hipStreamDestroy(streams[i].stream);
  }
  streams.resize(0);
}
